#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#include "hip/hip_runtime_api.h"

typedef unsigned int uint;

extern "C"
{
    void cudaInit()
    {
        int devID;

        // use device with highest Gflops/s
        devID = findCudaDevice();

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }
    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

    void copyArrayFromDevice(void *host, const void *device, int size)
    {
        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    }


    uint sumNumbers(uint *dNumbers, uint n)
    {
        // simple reduction from 1 to n
        thrust::device_ptr<uint> dp_numbers(dNumbers);
        return thrust::reduce(dp_numbers, dp_numbers + n);
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }
}
